#include "hip/hip_runtime.h"

/*
  By Ben Barsdell (2013)
  
  A simple GPU incoherent sum implementation.
*/

#include <cstdio>
#include <cstdlib>
#include <cstring> // For memcpy
#include <stdexcept>
#include <vector>
#include <string>
#include <iostream>
using std::cout;
using std::cerr;
using std::endl;
#include <fstream>
#include <sstream>
#include <iterator>
#include <cmath>

#include <errno.h>       // For errno
#include <sys/syscall.h> // For SYS_gettid

#include <dada_def.h>
#include <ascii_header.h>
#include <dada_cuda.h>

#include <thrust/device_vector.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>

//#include "aaplus/AA+.h" // Astronomical Algorithms C++ library (for coord conversions)

#include "dada_db2db.hpp"

int dada_bind_thread_to_core(int core)
{

  cpu_set_t set;
  pid_t tpid;

  CPU_ZERO(&set);
  CPU_SET(core, &set);
  tpid = syscall(SYS_gettid);

  if (sched_setaffinity(tpid, sizeof(cpu_set_t), &set) < 0) {
    fprintf(stderr, "failed to set cpu affinity: %s", strerror(errno));
    return -1;
  }

  CPU_ZERO(&set);
  if ( sched_getaffinity(tpid, sizeof(cpu_set_t), &set) < 0 ) {
    fprintf(stderr, "failed to get cpu affinity: %s", strerror(errno));
    return -1;
  }

  return 0;
}

template <typename T>
int sgn(T val) {
    return (T(0) < val) - (val < T(0));
}

inline __host__ __device__
float2 operator+(float2 a, float2 b) {
	return make_float2(a.x+b.x, a.y+b.y);
}

struct raw2stokesIV : public thrust::unary_function<char4,float2> {
	inline __host__ __device__
	float2 operator()(char4 val) const {
		float stokes_I = 0.f;
		stokes_I += val.x*val.x;
		stokes_I += val.y*val.y;
		stokes_I += val.z*val.z;
		stokes_I += val.w*val.w;
		
		float stokes_V = 0.f;
		stokes_V -= val.y*val.z;
		stokes_V += val.x*val.w;
		
		return make_float2(stokes_I, stokes_V);
	}
};

struct raw2XXYY : public thrust::unary_function<char4,float2> {
	inline __host__ __device__
	float2 operator()(char4 val) const {
		float xx = 0.f;
		xx += val.x*val.x;
		xx += val.y*val.y;
		float yy = 0.f;
		yy += val.z*val.z;
		yy += val.w*val.w;
		
		return make_float2(xx, yy);
	}
};

template<typename T>
struct divide_by : public thrust::unary_function<T,T> {
	T divisor;
	divide_by(T d) : divisor(d) {}
	inline __host__ __device__
	T operator()(T x) const {
		return x / divisor;
	}
};

class dbbeam : public dada_db2db {
	typedef char4  intype;
	typedef float2 outtype;
	
	int    m_gpu_device;
	double m_lat, m_lon;
	int    m_mode;
	float  m_max_aperture;
	bool   m_maintain_circular_aperture;
	size_t m_ntime, m_nchan, m_nstation, m_npol;
	size_t m_sample_offset;
	float3 m_pointing0, m_pointing90;
	float  m_lowfreq, m_df, m_dt;
	
	thrust::device_vector<intype>  m_d_in;
	thrust::device_vector<outtype> m_d_out;
	
protected:
	virtual void     onConnect(key_t in_key, key_t out_key) {
		// Register buffers as pinned memory
		dada_cuda_select_device(m_gpu_device);
		logInfo("dbbeam_gpu: Registering input buffer");
		dada_cuda_dbregister(this->hdu_in());
		logInfo("dbbeam_gpu: Registering output buffer");
		dada_cuda_dbregister(this->hdu_out());
	}
	virtual void     onDisconnect() {}
	// Return desired no. bytes per data read
	virtual uint64_t onHeader(uint64_t    header_size,
	                          const char* header_in,
	                          char*       header_out) {
		// Copy the in header to the out header
		memcpy(header_out, header_in, header_size);
		
		int ret;
		
		// Note: Datestamp format is "2013-06-16-03:46:53"
		char utc_start_str[32];
		ret = ascii_header_get(header_in, "UTC_START", "%s", utc_start_str);
		if( ret < 0 ) {
			throw std::runtime_error("Missing header entry UTC_START");
		}
		//int year, month, day, hour, minute;
		//float second;
		/*
		  ret = sscanf(utc_start_str, "%i-%02i-%02i-%02i:%02i:%f",
		  &year, &month, &day, &hour, &minute, &second);
		  if( ret != 6 ) {
		  cerr << "UTC_START = " << utc_start_str << endl;
		  throw std::runtime_error("Could not parse UTC_START");
		  }
		*/
		tm time;
		if( !strptime(utc_start_str, "%Y-%m-%d-%H:%M:%S", &time) ) {
			cerr << "UTC_START = " << utc_start_str << endl;
			throw std::runtime_error("Failed to parse UTC_START");
		}
		/*
		year   = time.tm_year+1900;
		month  = time.tm_mon+1;
		day    = time.tm_mday;
		hour   = time.tm_hour;
		minute = time.tm_min;
		second = time.tm_sec;
		double utc_start_jd = get_Julian_day(year, month, day, hour, minute, second);
		*/
		char ra_str[32];
		char dec_str[32];
		int  sign, deg;
		int   hour, minute;
		float second;
		ret = ascii_header_get(header_in, "RA", "%s", ra_str);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry RA"); }
		ret = sscanf(ra_str, "%i:%i:%f", &hour, &minute, &second);
		if( ret != 3 ) {
			cerr << "RA = " << ra_str << endl;
			throw std::runtime_error("Could not parse RA");
		}
		double ra = (second + 60*(minute + 60*hour)) * 360 / (24*60*60);
		
		ret = ascii_header_get(header_in, "DEC", "%s", dec_str);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry DEC"); }
		ret = sscanf(dec_str, "%i:%i:%f", &deg, &minute, &second);
		if( ret != 3 ) {
			cerr << "DEC = " << dec_str << endl;
			throw std::runtime_error("Could not parse DEC");
		}
		sign  = sgn(deg);
		deg   = abs(deg);
		double dec = sign * (second + 60*(minute + 60*deg)) / (60*60);
		
		ret = ascii_header_get(header_in, "TSAMP", "%f", &m_dt);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry TSAMP"); }
		m_dt *= 1e-6; // TSAMP is in units of us, we want seconds
		ret = ascii_header_get(header_in, "NCHAN", "%lu", &m_nchan);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry NCHAN"); }
		ret = ascii_header_get(header_in, "NSTATION", "%lu", &m_nstation);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry NSTATION"); }
		ret = ascii_header_get(header_in, "NPOL", "%lu", &m_npol);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry NPOL"); }
		ret = ascii_header_get(header_in, "LOWFREQ", "%f", &m_lowfreq);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry LOWFREQ"); }
		ret = ascii_header_get(header_in, "CHAN_WIDTH", "%f", &m_df);
		if( ret < 0 ) { throw std::runtime_error("Missing/invalid header entry CHAN_WIDTH"); }
		
		m_ntime = this->bufsize_out() / (m_nchan*m_npol*sizeof(outtype));
		
		cout << "UTC      = " << utc_start_str << endl;
		//cout << "UTC JD   = " << utc_start_jd << endl;
		cout << "dt       = " << m_dt << endl;
		cout << "ntime    = " << m_ntime << endl;
		cout << "nchan    = " << m_nchan << endl;
		cout << "nstation = " << m_nstation << endl;
		cout << "npol     = " << m_npol << endl;
		cout << "lowfreq  = " << m_lowfreq << endl;
		cout << "df       = " << m_df << endl;
		cout << "ra       = " << ra << endl;
		cout << "dec      = " << dec << endl;
		
		// Update (some) parameter(s) in the header
		uint64_t outsize      = this->bufsize_out();//m_ntime*m_nchan*m_npol*sizeof(outtype);
		uint64_t max_filesize = 2ull*1024*1024*1024;
		uint64_t bytes_per_second = (max_filesize-header_size) / (outsize * 10) * outsize;
		if( ascii_header_set(header_out, "NBIT", "%d", 32) < 0 ) {
			logInfo("dbbeam: Failed to set NBIT 32 in header_out");
		}
		if( ascii_header_set(header_out, "BYTES_PER_SECOND", "%i", bytes_per_second) < 0 ) {
			logInfo("dbbeam: Failed to set BYTES_PER_SECOND in header_out");
		}
		if( ascii_header_set(header_out, "DATA_ORDER", "%s", "time_chan_pol_cpx_f32") < 0 ) {
			logInfo("dbbeam: Failed to set DATA_ORDER in header_out");
		}
		if( ascii_header_set(header_out, "SOURCE", "%s", "TARGET") < 0 ) {
			logInfo("dbbeam: Failed to set SOURCE in header_out");
		}
		if( ascii_header_set(header_out, "MODE", "%s", "SINGLE_BEAM") < 0 ) {
			logInfo("dbbeam: Failed to set MODE in header_out");
		}
		
		m_d_in.reserve(m_ntime*m_nchan*m_nstation);
		m_d_out.resize(m_ntime*m_nchan);
		
		size_t bytes_per_read = m_ntime*m_nchan*m_nstation*sizeof(intype);
		cout << "bytes_per_read = " << bytes_per_read << endl;
		return bytes_per_read;
	}
	
	uint64_t beamform_incoherent(const intype*  __restrict__ in,
	                             outtype* __restrict__ out) {
		size_t count = m_ntime*m_nchan*m_nstation;
		m_d_in.assign(in, in + count);
		
		using thrust::make_transform_iterator;
		using thrust::make_counting_iterator;
		using thrust::make_discard_iterator;
		
		thrust::reduce_by_key(make_transform_iterator(make_counting_iterator<uint>(0),
		                                              divide_by<uint>(m_nstation)),
		                      make_transform_iterator(make_counting_iterator<uint>(0),
		                                              divide_by<uint>(m_nstation))+count,
		                      make_transform_iterator(m_d_in.begin(),
		                                              //raw2stokesIV()),
		                                              raw2XXYY()),
		                      make_discard_iterator(),
		                      m_d_out.begin());
		
		thrust::copy(m_d_out.begin(), m_d_out.end(),
		             out);
		
		// TODO: This is only half as much data as in the coherent implementation
		size_t bytes_written = m_ntime*m_nchan*sizeof(outtype);
		return bytes_written;
	}
	
	// Return no. bytes written
	virtual uint64_t onData(uint64_t    in_size,
	                        const char* data_in,
	                        char*       data_out) {
		const intype* __restrict__ in  = (const intype*)data_in;
		outtype*      __restrict__ out =      (outtype*)data_out;
		
		switch( m_mode ) {
		case BF_MODE_INCOHERENT: return beamform_incoherent(in, out);
		//case BF_MODE_COHERENT:   return beamform_coherent(in, out);
		default: throw std::runtime_error("Invalid beamforming mode");
		}
	}
	
public:
	enum { BF_MODE_INCOHERENT, BF_MODE_COHERENT };
	
	dbbeam(multilog_t* log, int verbose, int gpu_device,
	       double lat, double lon,
	       int mode=BF_MODE_COHERENT,
	       float max_aperture=1e99, bool maintain_circular_aperture=false)
		: dada_db2db(log, verbose),
		  m_gpu_device(gpu_device),
		  m_lat(lat), m_lon(lon),
		  m_mode(mode),
		  m_max_aperture(max_aperture),
		  m_maintain_circular_aperture(maintain_circular_aperture) {
		
		// Give the CPU a rest while the GPU kernel is running
		hipSetDeviceFlags(hipDeviceScheduleYield);
		
		/*
		hipError_t error = hipSetDevice(gpu_device);
		if( error != hipSuccess ) {
			throw std::runtime_error(hipGetErrorString(error));
		}
		*/
	}
	virtual ~dbbeam() {}
};

bool parse_arg_typed(int& x)                { return sscanf(optarg, "%i", &x) == 1; }
bool parse_arg_typed(unsigned& x)           { return sscanf(optarg, "%u", &x) == 1; }
bool parse_arg_typed(long long& x)          { return sscanf(optarg, "%lli", &x) == 1; }
bool parse_arg_typed(unsigned long long& x) { return sscanf(optarg, "%llu", &x) == 1; }
bool parse_arg_typed(float& x)              { return sscanf(optarg, "%f", &x) == 1; }
bool parse_arg_typed(std::string& x)        { x = optarg; return true; }
template<typename T>
bool parse_arg(char c, T& x) {
	if( !optarg ) {
		cerr << "ERROR: -" << c << " flag requires an argument" << endl;
		return false;
	}
	else if( !parse_arg_typed(x) ) {
		cerr << "ERROR: Could not parse -" << c << " " << optarg << endl;
		return false;
	}
	else {
		return true;
	}
}
/*
int load_stands(std::string          filename,
                std::vector<float3>& stands_xyz,
                std::vector<float>&  delays_low,
                std::vector<float>&  delays_high) {
	
	// *** TODO: IMPORTANT: Must convert into LEDA correlator order
	
	std::ifstream standstream(filename.c_str());
	if( !standstream ) {
		cerr << "ERROR: Failed to open stands file " << filename << endl;
		return -1;
	}
	while( standstream.good() ) {
		std::string line;
		std::getline(standstream, line);
		if( line.length() == 0 || line[0] == '#' ) {
			continue;
		}
		std::stringstream ss;
		int idx;
		float x, y, z, da_low, da_high, db_low, db_high;
		ss >> idx >> x >> y >> z >> da_low >> da_high >> db_low >> db_high;
		// TODO: This assumes the stands are perfectly ordered in the file
		//         Using std::map is probably the best way to do it properly
		stands_xyz.push_back(float3(x,y,z));
		delays_low.push_back(da_low);
		delays_low.push_back(db_low);
		delays_high.push_back(da_high);
		delays_high.push_back(db_high);
	}
	return 0;
}
*/
void print_usage() {
	cout << 
		"dbbeam [options] -- lat lon in_key out_key\n"
		" lat/lon      Observatory latitude and longitude as decimals\n"
		" -d gpu_idx   Index of GPU to use\n"
		" -s standfile Stand data file to use [stands.txt]\n"
		" -i           Incoherent sum only\n"
		" -a aperture  Max aperture (dist. from centre of array) [1e99]\n"
		" -b           Maintain circular aperture (at cost of area)\n"
		" -c core      Bind process to CPU core\n"
		" -v           Increase verbosity\n"
		" -q           Decrease verbosity\n"
		" -h           Print usage\n" << endl;
}

int main(int argc, char* argv[])
{
	// TODO: Consider reading this (also low/highfreq) from an env var
	int         gpu_idx      = 0;
	std::string standfile    = "stands.txt";
	bool        incoherent   = false;
	float       max_aperture = 1e99;
	int         circular     = 0;
	int         core         = -1;
	int         verbose      = 0;
	float       lat          = 0;
	float       lon          = 0;
	key_t       in_key       = 0;
	key_t       out_key      = 0;
	multilog_t* log          = 0;
	
	int arg = 0;
	while( (arg = getopt(argc,argv,"d:s:ia:bc:hvq")) != -1 ) {
		switch( arg ) {
		case 'd': if( !parse_arg('d', gpu_idx) ) return -1; break;
		case 's': if( !parse_arg('s', standfile) ) return -1; break;
		case 'i': incoherent = true; break;
		case 'a': if( !parse_arg('a', max_aperture) ) return -1; break;
		case 'b': ++circular; break;
		case 'c': if( !parse_arg('c', core) ) return -1; break;
		case 'h': print_usage(); return 0;
		case 'v': ++verbose; break;
		case 'q': --verbose; break;
		default: cerr << "WARNING: Unexpected flag -" << arg << endl; break;
		}
	}
	int num_args = argc - optind;
	if( num_args != 4 ) {
		cerr << "ERROR: Expected exactly 4 required args, got " << num_args << endl;
		print_usage();
		return -1;
	}
	if( sscanf(argv[optind+0], "%f", &lat) != 1 ) {
		cerr << "ERROR: Could not parse latitude from "
		     << argv[optind+0] << endl;
		return -1;
	}
	if( sscanf(argv[optind+1], "%f", &lon) != 1 ) {
		cerr << "ERROR: Could not parse longitude from "
		     << argv[optind+1] << endl;
		return -1;
	}
	unsigned int tmp;
	if( sscanf(argv[optind+2], "%x", &tmp) != 1 ) {
		cerr << "ERROR: Could not parse buffer key from "
		     << argv[optind+2] << endl;
		return -1;
	}
	in_key = tmp;
	if( sscanf(argv[optind+3], "%x", &tmp) != 1 ) {
		cerr << "ERROR: Could not parse buffer key from "
		     << argv[optind+3] << endl;
		return -1;
	}
	out_key = tmp;
	
	if( verbose >= 1 ) {
		cout << "GPU idx    = " << gpu_idx << endl;
		cout << "Latitude   = " << lat << endl;
		cout << "Longitude  = " << lon << endl;
		cout << "In key     = " << std::hex << in_key << std::dec << endl;
		cout << "Out key    = " << std::hex << out_key << std::dec << endl;
		cout << "Incoherent = " << (incoherent ? "yes" : "no") << endl;
	}
	
	log = multilog_open("dbbeam", 0);
	multilog_add(log, stderr);
	
	if( core >= 0 ) {
		if( dada_bind_thread_to_core(core) < 0 ) {
			cerr << "WARNING: Failed to bind to core " << core << endl;
		}
		if( verbose >= 1 ) {
			cout << "Process bound to core " << core << endl;
		}
	}
	
	int mode;
	if( incoherent ) {
		mode = dbbeam::BF_MODE_INCOHERENT;
	}
	else {
		mode = dbbeam::BF_MODE_COHERENT;
	}
	
	dbbeam ctx(log, verbose, gpu_idx, lat, lon, mode, max_aperture, circular);
	ctx.connect(in_key, out_key);
	ctx.run();
	ctx.disconnect();
	
	return 0;
}
