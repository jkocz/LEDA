#include "hip/hip_runtime.h"
// -*- c++ -*-

#include "dada_cuda.h"
#include "dada_client.h"
#include "dada_hdu.h"
#include "dada_def.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"
#include "daemon.h"

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <errno.h>
#include <fcntl.h>
#include <assert.h>
#include <math.h>
#include <complex>
#include <limits.h>
#include <omp.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <inttypes.h>

/*
  Data ordering for input vectors is (running from slowest to fastest)
  [time][channel][station][polarization][complexity]

  actual data:
  we will have dropped some packets (udp)
  this will have to be compensated for in the unpacker. 
  - insert zeros/random repeat random sample?

  [frequency][station][pol][complex]

  Output matrix has ordering
  [channel][station][station][polarization][polarization][complexity]
*/

// uncomment to use 8-bit fixed point, comment out for 32-bit floating point

#define FIXED_POINT

// set the data type accordingly
#ifndef FIXED_POINT
typedef std::complex<float> ComplexInput;
#define COMPLEX_INPUT float2
#define SCALE 1.0f // no rescale required for FP32
#else
typedef std::complex<char> ComplexInput;
#define COMPLEX_INPUT char2 
#define SCALE 16129.0f // need to rescale result 
//#define SCALE 1.0f
#endif

#define REGISTER_TILE_TRIANGULAR_ORDER 3000
#define MATRIX_ORDER REGISTER_TILE_TRIANGULAR_ORDER

// size = freq * time * station * pol *sizeof(ComplexInput)
#define GBYTE (1024llu*1024llu*1024llu) 

//#define NFREQUENCY 814ll // num freq channels
//#define NFREQUENCY 40ll // num freq channels
#define NFREQUENCY 600ll // num freq channels
//#define NFREQUENCY 814ll // num freq channels
//#define NFREQUENCY 52ll // num freq channels
#define NPOL 2
//#define NSTATION 16ll
#define NSTATION 32ll
//#define NSTATION 256ll
#define NTIME 8192ll //SAMPLES / NFREQUENCY
//#define NTIME 24000ll //SAMPLES / NFREQUENCY
//#define NSTATION 16ll
//#define NSTATION 256ll
//#define NTIME 8192ll //SAMPLES / NFREQUENCY

//#define SIGNAL_SIZE GBYTE
#define TEST_BYTE (NFREQUENCY*NTIME*NSTATION*NPOL*sizeof(ComplexInput))
#define SIGNAL_SIZE TEST_BYTE
#define SAMPLES SIGNAL_SIZE / (NSTATION*NPOL*sizeof(ComplexInput))

#define NBASELINE ((NSTATION+1)*(NSTATION/2))
#define NDIM 2

#define NTIME_PIPE 1024
//#define NTIME_PIPE 4096

#define PIPE_LENGTH NTIME / NTIME_PIPE


// how many pulsars are we binning for (Not implemented yet)
#define NPULSAR 0

// whether we are writing the matrix back to device memory (used for benchmarking)
int writeMatrix = 1;
// this must be enabled for this option to work though, slightly hurts performance
//#define WRITE_OPTION 

/* 
  enable this option to receive data way the dada key
  system. Otherwise, random data will be generated locally
  and used.
*/
void leda_dbgpu_cleanup (dada_hdu_t * hdu_in, dada_hdu_t * hdu_out, multilog_t * log);
int dada_bind_thread_to_core (int core);


#define FROM_CPU 0
#define FROM_DADA 1

typedef std::complex<float> Complex;

Complex convert(const ComplexInput &b) {
  return Complex(real(b), imag(b));
}

int dada_bind_thread_to_core(int core)
{

  cpu_set_t set;
  pid_t tpid;

  CPU_ZERO(&set);
  CPU_SET(core, &set);
  tpid = syscall(SYS_gettid);

  if (sched_setaffinity(tpid, sizeof(cpu_set_t), &set) < 0) {
    fprintf(stderr, "failed to set cpu affinity: %s", strerror(errno));
    return -1;
  }

  CPU_ZERO(&set);
  if ( sched_getaffinity(tpid, sizeof(cpu_set_t), &set) < 0 ) {
    fprintf(stderr, "failed to get cpu affinity: %s", strerror(errno));
    return -1;
  }

  return 0;
}

/* 
   this file shouldn't need an output key, as it will write 
   the information to hd after unloading the gpu

*/

void usage()
{
  fprintf (stdout,
           "leda_dbgpu [options] in_key out_key\n"
           " -c core   bind process to CPU core\n"
           " -g dev    use specified cuda device [default 0]\n"
           " -v        verbose mode\n"
           " -h        print usage\n");
}

#include "cuda_xengine.cu"
#include "omp_xengine.cc"
#include "cpu_util.cc"

int main (int argc, char **argv)
{

  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in = 0;
  dada_hdu_t* hdu_out = 0;

  /* DADA Logger */
  multilog_t* log = 0;

  /* Flag set in daemon mode */
  //char daemon = 0;

  /* Flag set in verbose mode */
  char verbose = 0;

  /* CUDA device to use */
  int device = 0;

  int core = -1;

  // input data block HDU key
  key_t in_key = 0;

  // output data block HDU key
  key_t out_key = 0;

  int arg = 0;

  while ((arg=getopt(argc,argv,"c:g:hv")) != -1)
  {
    switch (arg)
    {
      case 'c':
	if (optarg)
	{
	  core = atoi(optarg);
	  break;
	}
	else
	{
	  fprintf (stderr, "ERROR: -c flag requires argument\n");
	  return EXIT_FAILURE;
	}

      case 'g':
        device = atoi(optarg);
        break;

      case 'h':
        usage();
        return EXIT_SUCCESS;

      case 'v':
        verbose++;
        break;
    }
  }

  int num_args = argc-optind;

  if (num_args != 2)
  {
    usage();
    return EXIT_FAILURE;
  }

#if (FROM_DADA)      
  if (verbose)
    fprintf (stderr, "leda_dbgpu: parsing input key=%s\n", argv[optind]);
  if (sscanf (argv[optind], "%x", &in_key) != 1)
  {
    fprintf (stderr, "leda_dbgpu: could not parse in key from %s\n", argv[optind]);
    return EXIT_FAILURE;
  }

  if (verbose)
    fprintf (stderr, "leda_dbgpu: parsing output key=%s\n", argv[optind+1]);
  if (sscanf (argv[optind+1], "%x", &out_key) != 1)
  {
    fprintf (stderr, "leda_dbgpu: could not parse out key from %s\n", argv[optind+1]);
    return EXIT_FAILURE;
  }

  log = multilog_open ("leda_dbgpu", 0);

  multilog_add (log, stderr);

  if (verbose)
    multilog (log, LOG_INFO, "leda_dbgpu: creating in hdu\n");

  // open connection to the in/read DB
  hdu_in  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0)
    return EXIT_FAILURE;
  if (dada_hdu_lock_read (hdu_in) < 0)
    return EXIT_FAILURE;

  // open connection to the out/write DB
  hdu_out = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0)
  { 
    leda_dbgpu_cleanup (hdu_in, hdu_out, log);
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_write(hdu_out) < 0)
  {
    leda_dbgpu_cleanup (hdu_in, hdu_out, log);
    return EXIT_FAILURE;
  }
#endif
  
  if (core >= 0)
  {
    if (verbose)
      fprintf(stderr, "binding to core %d\n", core);
    if (dada_bind_thread_to_core(core) < 0)
      fprintf(stderr, "dbgpu: failed to bind to core %d\n", core);
  }

  //int64_t bytes_read=0;
  bool observation_complete=0;

  // AJ: I think there should be a NDIM in here 2 JK: also a sizeof(element) -> taken care of below
  uint64_t bytes=NFREQUENCY*NSTATION*NPOL*NTIME; 

  // bytes to read times sizeof(ComplexInput) 
  bytes *= 2;
  //fprintf(stderr, "sizeof(ComplexInput): %d\n",sizeof(ComplexInput));

  fprintf(stderr, "main: bytes_to_read=%llu\n", bytes);
  //fprintf(stderr, "main: bytes_to_read=%"PRIu64"\n", bytes);

  int nstation = NSTATION;

#if FROM_DADA

  uint64_t header_size = 0;

  // read the header from the input HDU
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
  {
    multilog(log ,LOG_ERR, "main: could not read next header\n");
    leda_dbgpu_cleanup (hdu_in, hdu_out, log);
    return EXIT_FAILURE;
  }

  // read the number of statinon from the header
  if (ascii_header_get (header_in, "NSTATIONS", "%d", &nstation) != 1)
  {
    nstation = NSTATION;
    multilog(log, LOG_WARNING, "Header with no nstation. Setting to %d\n", NSTATION);
  }

  // now write the output DADA header
  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
  {
    multilog(log, LOG_ERR, "could not get next header block [output]\n");
    leda_dbgpu_cleanup (hdu_in, hdu_out, log);
    return EXIT_FAILURE;
  }

  // copy the in header to the out header
  memcpy (header_out, header_in, header_size);

  // need to change some DADA parameters
  if (ascii_header_set (header_out, "NBIT", "%d", 32) < 0)
    multilog(log, LOG_WARNING, "failed to set NBIT 32 in header_out\n");

  // mark the input header as cleared
  if (ipcbuf_mark_cleared (hdu_in->header_block) < 0)
  {
    multilog (log, LOG_ERR, "could not mark header block cleared [input]\n");
    leda_dbgpu_cleanup (hdu_in, hdu_out, log);
    return EXIT_FAILURE;
  }

  // mark the output header buffer as filled
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
  {
    multilog (log, LOG_ERR, "could not mark header block filled [output]\n");
    leda_dbgpu_cleanup (hdu_in, hdu_out, log);
    return EXIT_FAILURE;
  }

#endif 

  printf("Correlating %llu stations with %llu signals, with %llu channels and integration length %llu\n",
	 NSTATION, SAMPLES, NFREQUENCY, NTIME);
    
  //unsigned long long vecLength = NFREQUENCY * NTIME * NSTATION * NPOL;
    
  // perform host memory allocation
  //int packedMatLength = NFREQUENCY * ((NSTATION+1)*(NSTATION/2)*NPOL*NPOL);
  
  // allocate the GPU X-engine memory
  
  // int64_t bytes_read = ipcio_read(hdu->data_block, (char*)buffer, bytes);
  ComplexInput *array_h = 0;
  Complex *cuda_matrix_h = 0;
  Complex *cuda_matrix_h_avg = 0;
  xInit(&array_h, &cuda_matrix_h, NSTATION, device);

#if (FROM_CPU)
  random_complex(array_h, vecLength);
#endif

  // register the data_block buffers with cuda_host_register
  dada_cuda_dbregister (hdu_in);

  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  uint64_t bytes_to_read;
  uint64_t block_id;
  char *   block;
  uint64_t ibyte;
  int      avg_index = 0;
#if (FROM_CPU)
  int      fd;
  char     filename [50];
#endif
  //uint64_t matrix_index=0;
  int      bytes_to_write = 0;
  uint64_t bytes_written=0;
  uint64_t written=0;

  matLength = NFREQUENCY * ((Nstation/2+1)*(Nstation/4)*NPOL*NPOL*4);// * (NPULSAR + 1);
  cuda_matrix_h_avg = (Complex *) malloc (matLength*sizeof(Complex));
  bytes_to_write = matLength*sizeof(Complex);

  while (!observation_complete)
  {
    
    // open a DADA block
    block = ipcio_open_block_read (hdu_in->data_block, &bytes_to_read, &block_id); 
    if (verbose)
    	multilog(log, LOG_INFO, "main: opened block %llu which contains %llu bytes\n", block_id, bytes_to_read);

    for (ibyte=0; ibyte < bytes_to_read; ibyte += bytes)
    {
      if (verbose)
      	multilog(log, LOG_INFO, "main: [%llu] ibyte=%llu bytes_to_read=%llu bytes=%llu\n", block_id, ibyte, bytes_to_read, bytes);

      // can cudaXengine handle non full buffer? probably not...
      if (ibyte + bytes > bytes_to_read)
        multilog(log, LOG_INFO, "main: skipping cudaXEngine as non full block\n");
      else
        cudaXengine(cuda_matrix_h,  (ComplexInput *) block);

      //for (matrix_index =0; matrix_index < matLength; matrix_index+=1) 
	//      fprintf(stdout, "block[%d]: %d\n",matrix_index,block[matrix_index]); 

      // increment the block pointer by the gulp amount (in bytes)
      block += bytes;

      //multilog(log, LOG_INFO, "main: xengine complete\n");

      //for (int matrix_index =0; matrix_index < matLength; matrix_index+=1) 
        //      fprintf(stdout, "cuda_matrix_h[%d]: %f + %fi\n",matrix_index,real(cuda_matrix_h[matrix_index]),imag(cuda_matrix_h[matrix_index])); 

	for (int matrix_index = 0; matrix_index < matLength; matrix_index++)
	 cuda_matrix_h_avg[matrix_index] += cuda_matrix_h[matrix_index];
//	for (int matrix_index = 0; matrix_index < matLength; matrix_index++)
//		cuda_matrix_h_avg[matrix_index] += cuda_matrix_h[matrix_index];

       if (avg_index < 25)
       {
          avg_index++;
	   //fprintf(stdout, "avg index %d\n",avg_index); 
       }
       else
       {
 	    avg_index=0;
#if FROM_DADA
           //if (verbose)
             multilog(log, LOG_INFO, "main: writing to datablock [output] %d bytes\n", bytes_to_write);
           //written = ipcio_write (hdu_out->data_block, (char *) cuda_matrix_h, bytes_to_write);
           written = ipcio_write (hdu_out->data_block, (char *) cuda_matrix_h_avg, bytes_to_write);
           if (written < bytes_to_write)
           {
             multilog(log, LOG_ERR, "main: failed to write all data to datablock [output]\n");
             leda_dbgpu_cleanup (hdu_in, hdu_out, log);
             return EXIT_FAILURE;
           }
           bytes_written += written;
           if (verbose)
             multilog(log, LOG_INFO, "main: write %llu bytes, %llu total\n", written, bytes_written);
	
#else
           //Complex *matrix_reorder = cuda_matrix_h;
           //reorderMatrix(matrix_reorder);
              
           //for (matrix_index = 0; matrix_index < matLength; matrix_index++)
		//fprintf(stdout, "outputMatrix[%d]: [%f] + [%f]i\n",matrix_index,real(matrix_reorder[matrix_index]),imag(matrix_reorder[matrix_index]));
           //for (matrix_index = 0; matrix_index < matLength; matrix_index++)
		//fprintf(stdout, "OrigMatrix[%d]: [%f] + [%f]i\n",matrix_index,real(cuda_matrix_h[matrix_index]),imag(cuda_matrix_h[matrix_index]));
           // print out matrix
           sprintf(filename, "%s_%llu", "utc_start", bytes_written); 
           fd = open (filename,O_WRONLY|O_CREAT, S_IWRITE);
           written = write(fd,cuda_matrix_h, bytes_to_write);
	   bytes_written = bytes_written + written;
	   close(fd);
	   multilog(log, LOG_INFO, "main: writing complete, wrote: %d\n", bytes_to_write);
#endif
          	   
           memset(cuda_matrix_h_avg, '\0', matLength*sizeof(Complex));
       }

    }

    if (bytes_to_read < block_size)
      observation_complete = 1;

    ipcio_close_block_read (hdu_in->data_block, bytes_to_read);

    // check for end of data in the DADA block
    if (ipcbuf_eod((ipcbuf_t*) hdu_in->data_block))
    {
      multilog(log, LOG_INFO, "main: end of data reached, exiting\n");
      observation_complete = 1;
    }

#if (FROM_CPU)
    observation_complete = 1;
#endif

  }
  
  // free gpu memory
  xFree(array_h, cuda_matrix_h);  
  free(cuda_matrix_h_avg);

#if (FROM_DADA)
  leda_dbgpu_cleanup (hdu_in, hdu_out, log);
#endif
    
  return EXIT_SUCCESS;
}


void leda_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out, multilog_t * log)
{
  if (dada_hdu_unlock_read (in) < 0)
  {
    multilog(log, LOG_ERR, "could not unlock read on hdu_in\n");
  }
  dada_hdu_destroy (in);

  if (dada_hdu_unlock_write (out) < 0)
  {
    multilog(log, LOG_ERR, "could not unlock write on hdu_out\n");
  }
  dada_hdu_destroy (out);
}
